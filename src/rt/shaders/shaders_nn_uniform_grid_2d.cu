#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <optix.h>
#include <optix_device.h>

#include "geoms/distance.h"
#include "rt/launch_parameters.h"
#include "utils/derived_atomic_functions.h"
#include "utils/helpers.h"

enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };
// FLOAT_TYPE is defined by CMakeLists.txt
extern "C" __constant__ hd::details::LaunchParamsNNUniformGrid<FLOAT_TYPE, 2>
    params;

extern "C" __global__ void __intersection__nn_uniform_grid_2d() {
  auto point_a_id = optixGetPayload_0();
  auto n_hits = optixGetPayload_1();
  auto mbr_id = optixGetPrimitiveIndex();
  const auto& point_a = params.points_a[point_a_id];
  const auto& mbr_b = params.mbrs_b[mbr_id];
  auto radius = params.radius;
  auto min_dist2 = mbr_b.GetMinDist2(point_a);
  auto max_dist2 = mbr_b.GetMaxDist2(point_a);
  auto update_cmin2 = [](FLOAT_TYPE dist2) {
    FLOAT_TYPE cmin2;
    if (sizeof(FLOAT_TYPE) == sizeof(float)) {
      auto cmin2_storage = optixGetPayload_4();
      cmin2 = *reinterpret_cast<FLOAT_TYPE*>(&cmin2_storage);

      if (dist2 < cmin2) {
        cmin2 = dist2;
        cmin2_storage = *reinterpret_cast<unsigned int*>(&cmin2);
        optixSetPayload_4(cmin2_storage);
      }
    } else {
      uint2 cmin2_storage{optixGetPayload_4(), optixGetPayload_5()};
      hd::unpack64(cmin2_storage.x, cmin2_storage.y, &cmin2);

      if (dist2 < cmin2) {
        cmin2 = dist2;
        hd::pack64(&cmin2, cmin2_storage.x, cmin2_storage.y);
        optixSetPayload_4(cmin2_storage.x);
        optixSetPayload_5(cmin2_storage.y);
      }
    }
  };

  n_hits++;
  optixSetPayload_1(n_hits);
  auto begin_clk = optixGetPayload_3();

  // first hit;
  if (begin_clk == std::numeric_limits<unsigned int>::max()) {
    begin_clk = clock();
    optixSetPayload_3(begin_clk);
  }

  auto max_kcycles = params.max_kcycles;
  uint32_t past_kcycles = ((uint32_t) clock() - begin_clk) / 1000;

  if (past_kcycles > max_kcycles / 2) {
    optixSetPayload_3(0);
    optixReportIntersection(0, 0);  // return implicitly
  }

  // this box is out of search radius
  // This improves the performance by a lot
  if (params.prune && min_dist2 > radius * radius) {
    return;
  }

  // max dist is less than cmax, cannot produce a greater dist, so break
  // this almost does not improve performance
  if (params.prune && max_dist2 <= *params.cmax2) {
    update_cmin2(max_dist2);
    optixReportIntersection(0, 0);
  }

  auto begin = params.prefix_sum[mbr_id];
  auto end = params.prefix_sum[mbr_id + 1];

  for (auto offset = begin; offset < end; ++offset) {
    auto point_b_id = params.point_b_ids[offset];
    const auto& point_b = params.points_b[point_b_id];
    auto dist2 = hd::EuclideanDistance2(point_a, point_b);

    if (dist2 <= radius * radius) {
      update_cmin2(dist2);
    }

    if (params.eb && dist2 <= *params.cmax2) {
      optixSetPayload_2(optixGetPayload_2() + (offset - begin + 1));
      optixReportIntersection(0, 0); // return implicitly
    }
  }
  optixSetPayload_2(optixGetPayload_2() + end - begin);
}

extern "C" __global__ void __anyhit__nn_uniform_grid_2d() {
  optixTerminateRay();
}

extern "C" __global__ void __raygen__nn_uniform_grid_2d() {
  const auto& in_queue = params.in_queue;
  float tmin = 0;
  float tmax = FLT_MIN;

  for (auto i = optixGetLaunchIndex().x; i < in_queue.size();
       i += optixGetLaunchDimensions().x) {
    unsigned int point_id_a = in_queue[i];
    const auto& point_a = params.points_a[point_id_a];

    float3 origin;
    origin.x = point_a.x;
    origin.y = point_a.y;
    origin.z = 0;
    float3 dir = {0, 0, 1};

    auto cmin2 = std::numeric_limits<FLOAT_TYPE>::max();
    unsigned int n_hits = 0;
    unsigned int n_compared_pairs = 0;
    // max: unset, 0: timeout
    unsigned int begin_cycle = std::numeric_limits<unsigned int>::max();

    if (sizeof(FLOAT_TYPE) == sizeof(float)) {
      auto cmin2_storage = *reinterpret_cast<unsigned int*>(&cmin2);
      optixTrace(
          params.handle, origin, dir, tmin, tmax, 0, OptixVisibilityMask(255),
          OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
          SURFACE_RAY_TYPE,     // SBT offset
          RAY_TYPE_COUNT,       // SBT stride
          SURFACE_RAY_TYPE,     // missSBTIndex
          point_id_a, n_hits, n_compared_pairs, begin_cycle, cmin2_storage);
      cmin2 = *reinterpret_cast<FLOAT_TYPE*>(&cmin2_storage);
    } else {
      uint2 cmin2_storage;
      hd::pack64(&cmin2, cmin2_storage.x, cmin2_storage.y);
      optixTrace(params.handle, origin, dir, tmin, tmax, 0,
                 OptixVisibilityMask(255),
                 OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
                 SURFACE_RAY_TYPE,     // SBT offset
                 RAY_TYPE_COUNT,       // SBT stride
                 SURFACE_RAY_TYPE,     // missSBTIndex
                 point_id_a, n_hits, n_compared_pairs, begin_cycle,
                 cmin2_storage.x, cmin2_storage.y);
      hd::unpack64(cmin2_storage.x, cmin2_storage.y, &cmin2);
    }

    if (params.hit_counters != nullptr) {
      params.hit_counters[point_id_a] += n_hits;
    }

    if (params.point_counters != nullptr) {
      params.point_counters[point_id_a] += n_compared_pairs;
    }

    if (begin_cycle == 0) {  // timeout
      if (params.term_queue.capacity()) {
        params.term_queue.Append(point_id_a);
      }
    } else {
      if (cmin2 != std::numeric_limits<FLOAT_TYPE>::max()) {
        atomicMax(params.cmax2, cmin2);
      } else {
        if (params.miss_queue.capacity()) {
          params.miss_queue.Append(point_id_a);
        }
      }
    }
  }
}
