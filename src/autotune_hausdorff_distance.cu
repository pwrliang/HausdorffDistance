#include "hip/hip_runtime.h"
#include <glog/logging.h>

#include <algorithm>  // For std::shuffle
#include <chrono>
#include <cstdio>
#include <ctime>
#include <iomanip>
#include <iostream>
#include <random>  // For random number generators
#include <sstream>

#include "hausdorff_distance.h"
#include "hd_impl/hausdorff_distance_early_break.h"
#include "hd_impl/hausdorff_distance_hybrid.h"
#include "loader.h"
#include "loaders/img_loader.h"
#include "move_points.h"
#include "run_config.h"
#include "running_stats.h"
#include "utils/stopwatch.h"
#include "utils/type_traits.h"

namespace hd {
template <typename COORD_T, int N_DIMS>
void AutoTuneHausdorffDistanceImpl(const RunConfig& config);

inline std::string get_current_datetime_string() {
  auto now = std::chrono::system_clock::now();
  std::time_t time_now = std::chrono::system_clock::to_time_t(now);

  std::tm local_tm;
#if defined(_MSC_VER)
  localtime_s(&local_tm, &time_now);  // MSVC
#else
  localtime_r(&time_now, &local_tm);  // POSIX
#endif

  std::ostringstream oss;
  oss << std::put_time(&local_tm, "%Y-%m-%d %H:%M:%S");
  return oss.str();
}

void AutoTuneHausdorffDistance(const RunConfig& config) {
  if (config.is_double) {
    if (config.n_dims == 2) {
      // dist = RunHausdorffDistanceImpl<double, 2>(config);
    } else if (config.n_dims == 3) {
      // dist = RunHausdorffDistanceImpl<double, 3>(config);
    }
  } else {
    if (config.n_dims == 2) {
      // AutoTuneHausdorffDistanceImpl<float, 2>(config);
    } else if (config.n_dims == 3) {
      AutoTuneHausdorffDistanceImpl<float, 3>(config);
    }
  }
}

template <typename COORD_T, int N_DIMS>
void AutoTuneHausdorffDistanceImpl(const RunConfig& config) {
  using point_t = typename cuda_vec<COORD_T, N_DIMS>::type;
  using mbr_t = MbrTypeFromPoint<point_t>;
  std::vector<point_t> points_a, points_b;
  itk::Size<N_DIMS> img_size_a, img_size_b;
  Stream stream;

  switch (config.input_type) {
  case InputType::kImage: {
    points_a = LoadImage<COORD_T, N_DIMS>(config.input_file1, img_size_a,
                                          config.limit);
    points_b = LoadImage<COORD_T, N_DIMS>(config.input_file2, img_size_b,
                                          config.limit);
    break;
  }
  default: {
    points_a =
        LoadPoints<COORD_T, N_DIMS>(config.input_file1, config.serialize_folder,
                                    config.input_type, config.limit);
    points_b =
        LoadPoints<COORD_T, N_DIMS>(config.input_file2, config.serialize_folder,
                                    config.input_type, config.limit);
    break;
  }
  }
  CHECK_GT(points_a.size(), 0) << config.input_file1;
  CHECK_GT(points_b.size(), 0) << config.input_file2;

  LOG(INFO) << "Points A: " << points_a.size()
            << " Points B: " << points_b.size();
  if (config.move_offset != 0) {
    MovePoints(points_a, points_b, config.move_offset);
  }
  RunningStats& stats = RunningStats::instance();

  auto& json_gpu = stats.Log("GPU");
  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);  // Get current device ID
  hipGetDeviceProperties(&prop,
                          device);  // Get properties of the current device
  json_gpu["Device"] = device;
  json_gpu["name"] = prop.name;
  json_gpu["l2CacheSize"] = prop.l2CacheSize;
  json_gpu["multiProcessorCount"] = prop.multiProcessorCount;
  json_gpu["regsPerBlock"] = prop.regsPerBlock;
  json_gpu["maxThreadsPerBlock"] = prop.maxThreadsPerBlock;
  json_gpu["maxBlocksPerMultiProcessor"] = prop.maxBlocksPerMultiProcessor;
  json_gpu["regsPerMultiprocessor"] = prop.regsPerMultiprocessor;

  auto& json_input = stats.Log("Input");

  json_input["FileA"]["Path"] = config.input_file1;
  json_input["FileA"]["NumPoints"] = points_a.size();
  json_input["FileB"]["Path"] = config.input_file2;
  json_input["FileB"]["NumPoints"] = points_b.size();
  json_input["SerializationPrefix"] = config.serialize_folder;
  json_input["Limit"] = config.limit;
  json_input["NumDims"] = N_DIMS;
  json_input["Type"] = typeid(COORD_T) == typeid(float) ? "Float" : "Double";

  json_input["MoveOffset"] = config.move_offset;

  // Calculate MBR of points
  auto write_points_stats = [&](const std::string& key,
                                const thrust::device_vector<point_t>& points) {
    SharedValue<mbr_t> mbr;
    auto* p_mbr = mbr.data();

    UniformGrid<COORD_T, N_DIMS> stats_grid;

    mbr.set(stream.hip_stream(), mbr_t());
    thrust::for_each(thrust::cuda::par.on(stream.hip_stream()), points.begin(),
                     points.end(), [=] __device__(const point_t& p) mutable {
                       p_mbr->ExpandAtomic(p);
                     });
    auto h_mbr = mbr.get(stream.hip_stream());

    auto grid_size = stats_grid.CalculateGridResolution(
        h_mbr, points.size(), config.stats_n_points_cell);

    stats_grid.Init(grid_size, h_mbr);
    stats_grid.Insert(stream, points);
    stats_grid.ComputeHistogram();

    auto& dataset_stats_json = json_input[key];

    dataset_stats_json["Grid"] = stats_grid.GetStats();
    auto json_mbr = nlohmann::json::array();

    for (int dim = 0; dim < N_DIMS; ++dim) {
      json_mbr.push_back(
          {{"Lower", h_mbr.lower(dim)}, {"Upper", h_mbr.upper(dim)}});
    }
    dataset_stats_json["MBR"] = json_mbr;
    dataset_stats_json["Density"] = points.size() / h_mbr.get_volume();
    return h_mbr;
  };

  thrust::device_vector<point_t> d_points_a = points_a, d_points_b = points_b;

  mbr_t merged_mbr = write_points_stats("FileA", d_points_a);
  merged_mbr.Expand(write_points_stats("FileB", d_points_b));
  json_input["Density"] =
      (points_a.size() + points_b.size()) / merged_mbr.get_volume();

  auto n_combinations =
      config.n_points_cell_list.size() * config.sample_rate_list.size() *
      config.max_hit_list.size() * config.radius_step_list.size() *
      config.sort_rays_list.size() * config.fast_build_bvh_list.size() *
      config.rebuild_bvh_list.size();
  uint32_t n_progress = 0;
  uint32_t n_skips = 0;
  double best_running_time = std::numeric_limits<double>::max();

  using hd_impl_t = HausdorffDistanceHybrid<COORD_T, N_DIMS>;
  typename hd_impl_t::Config hd_config;

  std::string ptx_root = config.exec_path + "/ptx";

  hd_config.ptx_root = ptx_root.c_str();

  auto hausdorff_distance = std::make_unique<hd_impl_t>(hd_config);

  Stopwatch sw_begin;
  sw_begin.start();
  std::ostringstream oss;
  for (size_t i = 0; i < config.max_hit_list.size(); ++i) {
    if (i != 0)
      oss << ',';
    oss << config.max_hit_list[i];
  }
  auto s_max_hit_list = oss.str();

  for (auto n_points_cell : config.n_points_cell_list) {
    for (auto sample_rate : config.sample_rate_list) {
      for (auto radius_step : config.radius_step_list) {
        for (auto sort_rays : config.sort_rays_list) {
          for (auto fast_build_bvh : config.fast_build_bvh_list) {
            for (auto rebuild_bvh : config.rebuild_bvh_list) {
              CHECK_GT(n_points_cell, 0) << "Avg points / cell cannot be zero";
              CHECK_GT(radius_step, 1);
              CHECK_LE(sample_rate, 1);

              VLOG(1) << "N_points_cell = " << n_points_cell
                      << ", sample_rate = " << sample_rate
                      << ", max_hit_list = " << s_max_hit_list
                      << ", radius = " << radius_step
                      << ", sort_rays = " << sort_rays
                      << ", fast_build_bvh = " << fast_build_bvh
                      << ", rebuild_bvh = " << rebuild_bvh;

              char path[PATH_MAX];
              sprintf(path,
                      "%s_n_points_cell_%u_sample_rate_%.6f_max_hit_list_%s_"
                      "radius_step_%.2f_sort_rays_%d_fast_build_bvh_%d_rebuild_"
                      "bvh_%d.json",
                      config.json_file.c_str(), n_points_cell, sample_rate,
                      s_max_hit_list.c_str(), radius_step, sort_rays,
                      fast_build_bvh, rebuild_bvh);
              bool file_exists = access(path, R_OK) == 0;

              if (!config.json_file.empty() && !config.overwrite &&
                  file_exists) {
                n_skips++;
                continue;
              }

              stats.Log("DateTime", get_current_datetime_string());

              auto& json_run = stats.Log("Running");

              json_run.clear();
              json_run["StatsNumPointsPerCell"] = config.stats_n_points_cell;
              json_run["Seed"] = config.seed;
              json_run["SortRays"] = sort_rays;
              json_run["FastBuildBVH"] = fast_build_bvh;
              json_run["RebuildBVH"] = rebuild_bvh;
              json_run["RadiusStep"] = radius_step;
              json_run["SampleRate"] = sample_rate;
              json_run["MaxHitList"] = s_max_hit_list;
              json_run["NumPointsPerCell"] = n_points_cell;

              COORD_T dist = -1;

              hd_config.sort_rays = sort_rays;
              hd_config.fast_build = fast_build_bvh;
              hd_config.rebuild_bvh = rebuild_bvh;
              hd_config.radius_step = radius_step;
              hd_config.sample_rate = sample_rate;
              hd_config.n_points_cell = n_points_cell;

              hausdorff_distance->UpdateConfig(hd_config);

              double running_time = 0;

              for (int i = 0; i < config.repeat; i++) {
                auto& json_repeat = json_run["Repeat" + std::to_string(i)];
                json_run["Variant"] = "Hybrid";
                json_run["Execution"] = "GPU";
                dist = hausdorff_distance->CalculateDistance(
                    stream, d_points_a, d_points_b, config.max_hit_list);
                json_repeat = hausdorff_distance->get_stats();
                auto total_time = json_repeat.at("TotalTime").get<double>();
                running_time += total_time;
              }
              best_running_time =
                  std::min(best_running_time, running_time / config.repeat);
              json_run["AvgTime"] = running_time / config.repeat;
              LOG(INFO) << std::fixed << std::setprecision(2)
                        << "Avg Running Time " << running_time / config.repeat
                        << " ms";

              stats.Log("HDResult", dist);

              if (config.check) {
                using hd_reference_impl =
                    HausdorffDistanceEarlyBreak<COORD_T, N_DIMS>;
                auto& json_check = stats.Log("Check");
                typename hd_reference_impl::Config hd_config;

                hd_config.n_threads = std::thread::hardware_concurrency();
                auto hd_reference = std::make_unique<
                    HausdorffDistanceEarlyBreak<COORD_T, N_DIMS>>(hd_config);
                auto answer_dist =
                    hd_reference->CalculateDistance(points_a, points_b);
                auto diff = answer_dist - dist;

                json_check["HDAnswer"] = answer_dist;
                if (dist != answer_dist) {
                  LOG(ERROR) << std::fixed << std::setprecision(8)
                             << "Wrong HausdorffDistance. Result: " << dist
                             << " Answer: " << answer_dist << " Diff: " << diff;
                } else {
                  LOG(INFO) << "HausdorffDistance is checked";
                }
                json_check["Diff"] = diff;
                json_check["Pass"] = dist == answer_dist;
              }

              if (!config.json_file.empty()) {
                if (!file_exists || file_exists && config.overwrite) {
                  stats.Dump(path);
                } else {
                  LOG(WARNING) << "Skip writting to JSON file " << path;
                }
              }

              n_progress++;
              sw_begin.stop();

              LOG(INFO) << "Progress " << std::fixed << std::setprecision(2)
                        << (float) (n_progress + n_skips) / n_combinations * 100
                        << " % Remaining Time "
                        << sw_begin.ms() / n_progress *
                               (n_combinations - n_progress - n_skips) / 1000
                        << " s Best Performance " << best_running_time << " ms";
            }
          }
        }
      }
    }
  }
}
}  // namespace hd
